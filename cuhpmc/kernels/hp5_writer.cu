#include "hip/hip_runtime.h"
/* Copyright STIFTELSEN SINTEF 2012
 *
 * This file is part of the HPMC Library.
 *
 * Author(s): Christopher Dyken, <christopher.dyken@sintef.no>
 *
 * HPMC is free software: you can redistribute it and/or modify it under the
 * terms of the GNU General Public License as published by the Free Software
 * Foundation, either version 3 of the License, or (at your option) any later
 * version.
 *
 * HPMC is distributed in the hope that it will be useful, but WITHOUT ANY
 * WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
 * A PARTICULAR PURPOSE.  See the GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License along with
 * HPMC.  If not, see <http://www.gnu.org/licenses/>.
 */
#include <hip/hip_runtime.h>
#include <builtin_types.h>

namespace cuhpmc {

// constant mem size: 64kb, cache working set: 8kb.
// Count + pad :  1+3 elements :    16 bytes :    16 bytes
// Level 0     :    4 elements :    16 bytes :    32 bytes
// Level 1     :   20 elements :    80 bytes :   112 bytes
// Level 2     :  100 elements :   400 bytes :   512 bytes
// Level 3     :  500 elements :  2000 bytes :  2112 bytes
// Level 4     : 2500 elements : 10000 bytes : 12112 bytes
// Levels 0-2: 32*4*4=512 bytes :
// Level  3:

texture<uint4, 1, hipReadModeElementType> hp5_hp_tex;
__constant__ uint4 hp5_hp_const[528]; // = 2112/4
__constant__ uint  hp5_const_offsets[32];


template<bool use_texfetch,bool use_constmem>
__global__
void
dummy_writer( float* __restrict__       output_d,
              const uint4* __restrict__ hp5_d,
              const uint                triangles,
              const uint                max_level,
              const unsigned char*      field_d )
{
    uint vtx = 3*32*blockIdx.x + 32*threadIdx.y + threadIdx.x;
    if( vtx < 3*triangles ) {
        output_d[ 6*vtx + 0 ] = 0.001f*vtx;
        output_d[ 6*vtx + 1 ] = 0.f;
        output_d[ 6*vtx + 2 ] = 0.f;
        output_d[ 6*vtx + 3 ] = 0.001f*vtx;
        output_d[ 6*vtx + 4 ] = 0.f;
        output_d[ 6*vtx + 5 ] = 0.f;
    }
}

void
run_dummy_writer( float*                output_d,
                  const uint4*          hp5_pyramid_d,
                  const uint*           hp5_level_offsets_d,
                  const uint            hp5_size,
                  const uint            hp5_max_level,
                  const uint            triangles,
                  const unsigned char*  field_d,
                  const uint3           field_size,
                  hipStream_t          stream )
{
    bool use_constmem = true;
    bool use_texfetch = true;

    // Copy offsets to symbol
    hipMemcpyToSymbolAsync(HIP_SYMBOL( hp5_const_offsets),
                             hp5_level_offsets_d,
                             sizeof(uint)*32,
                             0,
                             hipMemcpyDeviceToDevice,
                             stream );
    // Copy top levels of hp if desired
    if( use_constmem ) {
        hipMemcpyToSymbolAsync(HIP_SYMBOL( hp5_hp_const),
                                 hp5_pyramid_d,
                                 528*sizeof(uint4),
                                 0,
                                 hipMemcpyDeviceToDevice,
                                 stream );
    }
    // Bind histopyramid as texture if desired
    if( use_texfetch ) {
        hipBindTexture( NULL,
                         hp5_hp_tex,
                         hp5_pyramid_d,
                         hipCreateChannelDesc( 32, 32, 32, 32,
                                                hipChannelFormatKindUnsigned ),
                         4*sizeof(uint)*hp5_size );
    }


    dim3 gs( (triangles+31)/32 );
    dim3 bs( 32, 3 );

    if( use_texfetch ) {
        if( use_constmem ) {
            dummy_writer<true,true><<<gs,bs,0,stream>>>( output_d,
                                                         hp5_pyramid_d,
                                                         triangles,
                                                         hp5_max_level,
                                                         field_d );
        }
        else {
            dummy_writer<true,false><<<gs,bs,0,stream>>>( output_d,
                                                          hp5_pyramid_d,
                                                          triangles,
                                                          hp5_max_level,
                                                          field_d );
        }
    }
    else {
        if( use_constmem ) {
            dummy_writer<false,true><<<gs,bs,0,stream>>>( output_d,
                                                          hp5_pyramid_d,
                                                          triangles,
                                                          hp5_max_level,
                                                          field_d );
        }
        else {
            dummy_writer<false,false><<<gs,bs,0,stream>>>( output_d,
                                                           hp5_pyramid_d,
                                                           triangles,
                                                           hp5_max_level,
                                                           field_d );
        }
    }
}



#if 0


{
    const uint ix = blockDim.x * blockIdx.x + threadIdx.x;
    if( ix < M ) {
        uint key = ix;
        uint pos = 0;
        int l=0;
        if( use_constmem ) {
            for(l=0; l<4; l++ ) {
                uint4 val = hp5_hp_const[ hp5_const_offsets[l] + pos ];
                pos *= 5;
                if( val.x <= key ) {
                    pos++;
                    key -=val.x;
                    if( val.y <= key ) {
                        pos++;
                        key-=val.y;
                        if( val.z <= key ) {
                            pos++;
                            key-=val.z;
                            if( val.w <= key ) {
                                pos++;
                                key-=val.w;
                            }
                        }
                    }
                }
            }
        }

        for(; l<max_level; l++) {
            uint4 val;
            if(use_texfetch) {
                val = tex1Dfetch( hp5_hp_tex, hp5_const_offsets[l] + pos );
            }
            else {
                val = d_hp[ hp5_const_offsets[l] + pos ];
            }
            pos *= 5;

            if( val.x <= key ) {
                pos++;
                key -=val.x;
                if( val.y <= key ) {
                    pos++;
                    key-=val.y;
                    if( val.z <= key ) {
                        pos++;
                        key-=val.z;
                        if( val.w <= key ) {
                            pos++;
                            key-=val.w;
                        }
                    }
                }
            }
        }
        d_result[ix]= d_input[pos];
    }
}


switch( path ) {
case 0:
    hp5_traverse<false,false><<<gs,bs>>>( d_hp5_output,
                                          d_hp5_hp,
                                          M,
                                          hp5_levels,
                                          d_input );
    break;
case 1:
    hp5_traverse<false,true><<<gs,bs>>>( d_hp5_output,
                                         d_hp5_hp,
                                         M,
                                         hp5_levels,
                                         d_input );
    break;
case 2:
    hp5_traverse<true,false><<<gs,bs>>>( d_hp5_output,
                                         d_hp5_hp,
                                         M,
                                         hp5_levels,
                                         d_input );
    break;
case 3:
    hp5_traverse<true,true><<<gs,bs>>>( d_hp5_output,
                                        d_hp5_hp,
                                        M,
                                        hp5_levels,
                                        d_input );
    break;
}
#endif
} // of namespace cuhpmc
