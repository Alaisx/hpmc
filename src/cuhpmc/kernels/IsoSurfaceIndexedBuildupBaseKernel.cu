#include "hip/hip_runtime.h"
/* Copyright STIFTELSEN SINTEF 2012
 *
 * This file is part of the HPMC Library.
 *
 * Author(s): Christopher Dyken, <christopher.dyken@sintef.no>
 *
 * HPMC is free software: you can redistribute it and/or modify it under the
 * terms of the GNU General Public License as published by the Free Software
 * Foundation, either version 3 of the License, or (at your option) any later
 * version.
 *
 * HPMC is distributed in the hope that it will be useful, but WITHOUT ANY
 * WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
 * A PARTICULAR PURPOSE.  See the GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License along with
 * HPMC.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <assert.h>
#include <iostream>
#include <stdexcept>
#include <cuhpmc/Constants.hpp>
#include <cuhpmc/FieldGlobalMemUChar.hpp>
#include <cuhpmc/IsoSurface.hpp>
#include <cuhpmc/IsoSurfaceIndexed.hpp>

namespace cuhpmc {

template<class T>
__device__
void
fetchFromField( uint&           bp0,                // Bit mask for slice 0
                uint&           bp1,                // Bit mask for slice 1
                uint&           bp2,                // Bit mask for slice 2
                uint&           bp3,                // Bit mask for slice 3
                uint&           bp4,                // Bit mask for slice 4
                uint&           bp5,                // Bit mask for slice 5
                const T*        field,              // Sample-adjusted field pointer
                const T*        field_end,          // Pointer to buffer end
                const size_t    field_row_pitch,
                const size_t    field_slice_pitch,
                const float     iso,
                const bool      no_check )
{
    const T* llfield = field;
    if( no_check ) {
        bp0 = (*llfield < iso) ? 1 : 0;
        llfield += field_slice_pitch;
        bp1 = (*llfield < iso) ? 1 : 0;
        llfield += field_slice_pitch;
        bp2 = (*llfield < iso) ? 1 : 0;
        llfield += field_slice_pitch;
        bp3 = (*llfield < iso) ? 1 : 0;
        llfield += field_slice_pitch;
        bp4 = (*llfield < iso) ? 1 : 0;
        llfield += field_slice_pitch;
        bp5 = (*llfield < iso) ? 1 : 0;
    }
    else {
        bp0 = ( llfield < field_end ) && (*llfield < iso) ? 1 : 0;
        llfield += field_slice_pitch;
        bp1 = ( llfield < field_end ) && (*llfield < iso) ? 1 : 0;
        llfield += field_slice_pitch;
        bp2 = ( llfield < field_end ) && (*llfield < iso) ? 1 : 0;
        llfield += field_slice_pitch;
        bp3 = ( llfield < field_end ) && (*llfield < iso) ? 1 : 0;
        llfield += field_slice_pitch;
        bp4 = ( llfield < field_end ) && (*llfield < iso) ? 1 : 0;
        llfield += field_slice_pitch;
        bp5 = ( llfield < field_end ) && (*llfield < iso) ? 1 : 0;
    }
}


template<class T>
struct hp5_buildup_base_indexed_triple_gb_args
{
    uint4* __restrict__             tri_pyramid_level_a_d;
    uint4* __restrict__             vtx_pyramid_level_a_d;
    uint4* __restrict__             tri_pyramid_level_b_d;
    uint4* __restrict__             vtx_pyramid_level_b_d;
    uint4* __restrict__             tri_pyramid_level_c_d;
    uint4* __restrict__             vtx_pyramid_level_c_d;
    uint*  __restrict__             tri_sideband_level_c_d;
    uint*  __restrict__             vtx_sideband_level_c_d;
    unsigned char* __restrict__     d_case;
    float                     iso;
    uint3                     cells;
    uint3                     chunks;
    const T* __restrict__                 field;
    const T* __restrict__           field_end;
    size_t                    field_row_pitch;
    size_t                    field_slice_pitch;
    const unsigned char*                  case_vtxtricnt;
};

template<class T>
__global__
void
__launch_bounds__( 160 )
hp5_buildup_base_indexed_triple_gb( hp5_buildup_base_indexed_triple_gb_args<T> a )
{
    __shared__ uint sb[800];
    __shared__ uint sh[801];

    const uint w  = threadIdx.x / 32;                                   // warp
    const uint wt = threadIdx.x % 32;                                   // thread-in-warp
    const uint sh_i = 160*w + 5*wt;                                     //
    const uint hp_b_o = 5*32*blockIdx.x + 32*w + wt;                    //
    const uint c_lix = 5*blockIdx.x + w;                                //


    const uint3 cp = make_uint3( 31*( c_lix % a.chunks.x ) + wt,          // field pos x
                                  5*( (c_lix/a.chunks.x) % a.chunks.y ),    // field pos y
                                  5*( (c_lix/a.chunks.x) / a.chunks.y ) );  // field pos.z
    const T* lfield = a.field +                                           // Field sample pointer
                      cp.x +
                      cp.y * a.field_row_pitch +
                      cp.z * a.field_slice_pitch;

    // Check if we are in danger of sampling outside the scalar field buffer
    bool no_check = lfield +
                      32 + 5*a.field_row_pitch + 5*a.field_slice_pitch < a.field_end;

    bool xmask = cp.x < a.cells.x;
    bool znocare = cp.z+5 < a.cells.z;

    // Fetch scalar field values and determine inside-outside for 5 slices
    uint bp0, bp1, bp2, bp3, bp4, bp5;

    fetchFromField( bp0, bp1, bp2, bp3, bp4, bp5,
                    lfield, a.field_end, a.field_row_pitch, a.field_slice_pitch,
                    a.iso,
                    no_check );

    for(uint q=0; q<5; q++) {
        // Move along y to build up masks
        uint bc0, bc1, bc2, bc3, bc4, bc5;
        fetchFromField( bc0, bc1, bc2, bc3, bc4, bc5,
                        lfield + (q+1)*a.field_row_pitch, a.field_end, a.field_row_pitch, a.field_slice_pitch,
                        a.iso, no_check );

        // Merge
        uint b0 = bp0 + (bc0<<2);
        uint b1 = bp1 + (bc1<<2);
        uint b2 = bp2 + (bc2<<2);
        uint b3 = bp3 + (bc3<<2);
        uint b4 = bp4 + (bc4<<2);
        uint b5 = bp5 + (bc5<<2);
        // Store for next iteration
        bp0 = bc0;
        bp1 = bc1;
        bp2 = bc2;
        bp3 = bc3;
        bp4 = bc4;
        bp5 = bc5;

        // build case
        uint m0_1 = b0 + (b1<<4);
        uint m1_1 = b1 + (b2<<4);
        uint m2_1 = b2 + (b3<<4);
        uint m3_1 = b3 + (b4<<4);
        uint m4_1 = b4 + (b5<<4);
        sh[ 0*160 + threadIdx.x ] = m0_1;
        sh[ 1*160 + threadIdx.x ] = m1_1;
        sh[ 2*160 + threadIdx.x ] = m2_1;
        sh[ 3*160 + threadIdx.x ] = m3_1;
        sh[ 4*160 + threadIdx.x ] = m4_1;

        uint ix_o_1 = 160*w + 32*q + wt;

        bool ymask = cp.y+q+1 < a.cells.y;
        uint sum;

        if( xmask && ymask && wt < 31 ) { // if-test needed to avoid syncthreads??
            m0_1 += (sh[ 0*160 + threadIdx.x + 1]<<1);
            m1_1 += (sh[ 1*160 + threadIdx.x + 1]<<1);
            m2_1 += (sh[ 2*160 + threadIdx.x + 1]<<1);
            m3_1 += (sh[ 3*160 + threadIdx.x + 1]<<1);
            m4_1 += (sh[ 4*160 + threadIdx.x + 1]<<1);

            // cnt_a_X = %00000000 0vv00ttt
            uint cnt_a_0 = a.case_vtxtricnt[ m0_1 ]; // Faster to fetch from glob. mem than tex.
            uint cnt_a_1 = a.case_vtxtricnt[ m1_1 ];
            uint cnt_a_2 = a.case_vtxtricnt[ m2_1 ];
            uint cnt_a_3 = a.case_vtxtricnt[ m3_1 ];
            uint cnt_a_4 = a.case_vtxtricnt[ m4_1 ];


            if( znocare ) {
                sum = cnt_a_0
                    + cnt_a_1
                    + cnt_a_2
                    + cnt_a_3
                    + cnt_a_4;
            }
            else {
                sum = (cp.z+0 < a.cells.z ? cnt_a_0 : 0) +
                      (cp.z+1 < a.cells.z ? cnt_a_1 : 0) +
                      (cp.z+2 < a.cells.z ? cnt_a_2 : 0) +
                      (cp.z+3 < a.cells.z ? cnt_a_3 : 0) +
                      (cp.z+4 < a.cells.z ? cnt_a_4 : 0);
            }
            // sum = %00000000 00000000 0000000v vvvttttt
            // sb  = %00000000 0000vvvv 00000000 000ttttt
            sb[ ix_o_1 ] = ((sum<<11)&0xf0000u) | (sum&0x1fu);
            if( sum > 0 ) {
                ((short1*)(a.tri_pyramid_level_a_d))[ 5*160*blockIdx.x + ix_o_1 ] =
                        make_short1( ((cnt_a_0 & 0xf)) |
                                     ((cnt_a_1 & 0xf)<<4) |
                                     ((cnt_a_2 & 0xf)<<8) |
                                     ((cnt_a_3 & 0xf)<<12) );

                        //   a.tri_pyramid_level_a_d[ 5*160*blockIdx.x + ix_o_1 ] = make_uint4( s0_1, s1_1, s2_1, s3_1 );
                a.d_case[ 5*(5*160*blockIdx.x + 160*w + 32*q + wt) + 0 ] = m0_1;
                a.d_case[ 5*(5*160*blockIdx.x + 160*w + 32*q + wt) + 1 ] = m1_1;
                a.d_case[ 5*(5*160*blockIdx.x + 160*w + 32*q + wt) + 2 ] = m2_1;
                a.d_case[ 5*(5*160*blockIdx.x + 160*w + 32*q + wt) + 3 ] = m3_1;
                a.d_case[ 5*(5*160*blockIdx.x + 160*w + 32*q + wt) + 4 ] = m4_1;
            }
        }
        else {
            sb[ ix_o_1 ] = 0;
        }
    }
    // second reduction
    uint cnt_b_0 = sb[ sh_i + 0 ];
    uint cnt_b_1 = sb[ sh_i + 1 ];
    uint cnt_b_2 = sb[ sh_i + 2 ];
    uint cnt_b_3 = sb[ sh_i + 3 ];
    uint cnt_b_4 = sb[ sh_i + 4 ];

    ((uchar4*)a.tri_pyramid_level_b_d)[ hp_b_o ] = make_uchar4( cnt_b_0,
                                                                cnt_b_1,
                                                                cnt_b_2,
                                                                cnt_b_3 );

    __syncthreads();
    // third reduction
    // sh = %00000000 0vvvvvvv 00000000 0ttttttt
    sh[ 32*w + wt ] = cnt_b_0
                    + cnt_b_1
                    + cnt_b_2
                    + cnt_b_3
                    + cnt_b_4;
    __syncthreads();
    if( w == 0 ) {
        uint cnt_c_0 = sh[5*wt+0];
        uint cnt_c_1 = sh[5*wt+1];
        uint cnt_c_2 = sh[5*wt+2];
        uint cnt_c_3 = sh[5*wt+3];
        uint cnt_c_4 = sh[5*wt+4];
        ((uchar4*)a.tri_pyramid_level_c_d)[ 32*blockIdx.x + wt ] = make_uchar4( cnt_c_0,
                                                                                cnt_c_1,
                                                                                cnt_c_2,
                                                                                cnt_c_3 );
        // sum = %0000000v vvvvvvvv 000000tt tttttttt
        uint sum = cnt_c_0
                 + cnt_c_1
                 + cnt_c_2
                 + cnt_c_3
                 + cnt_c_4;

        a.tri_sideband_level_c_d[ 32*blockIdx.x + wt ] = sum       & 0xffffu;
        a.vtx_sideband_level_c_d[ 32*blockIdx.x + wt ] = (sum>>16) & 0xffffu;
    }
}

void
IsoSurfaceIndexed::invokeBaseBuildup( hipStream_t stream )
{
    if( FieldGlobalMemUChar* field = dynamic_cast<FieldGlobalMemUChar*>( m_field ) ) {

        hp5_buildup_base_indexed_triple_gb_args<unsigned char> args;
        args.tri_pyramid_level_a_d  = m_triangle_pyramid_d + m_hp5_offsets[ m_hp5_levels-1 ];
        args.vtx_pyramid_level_a_d  = m_vertex_pyramid_d   + m_hp5_offsets[ m_hp5_levels-1 ];
        args.tri_pyramid_level_b_d  = m_triangle_pyramid_d + m_hp5_offsets[ m_hp5_levels-2 ];
        args.vtx_pyramid_level_b_d  = m_vertex_pyramid_d   + m_hp5_offsets[ m_hp5_levels-2 ];
        args.tri_pyramid_level_c_d  = m_triangle_pyramid_d + m_hp5_offsets[ m_hp5_levels-3 ];
        args.vtx_pyramid_level_c_d  = m_vertex_pyramid_d   + m_hp5_offsets[ m_hp5_levels-3 ];
        args.tri_sideband_level_c_d = m_triangle_sideband_d + m_hp5_offsets[ m_hp5_levels-3 ];
        args.vtx_sideband_level_c_d = m_vertex_sideband_d + m_hp5_offsets[ m_hp5_levels-3 ];
        args.d_case             = m_case_d;
        args.iso                = 256.f*m_iso;
        args.cells              = make_uint3( field->width()-1,
                                              field->height()-1,
                                              field->depth()-1 );
        args.chunks             = m_hp5_chunks;
        args.field              = field->fieldDev();
        args.field_end          = field->fieldDev() + field->width()*field->height()*field->depth();
        args.field_row_pitch    = field->width();
        args.field_slice_pitch  = field->width()*field->height();
        args.case_vtxtricnt     = m_constants->vertexTriangleCountDev() ;

        uint gs = (m_hp5_level_sizes[ m_hp5_levels-1 ]+3999)/4000;
        uint bs = 160;
        hp5_buildup_base_indexed_triple_gb<unsigned char><<<gs,bs,0, stream >>>( args );

    }
    else {
        throw std::runtime_error( "invokeBaseBuildup: unsupported field type" );
    }
}


} // of namespace cuhpmc
